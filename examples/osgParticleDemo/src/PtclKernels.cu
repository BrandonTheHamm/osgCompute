
#include <hip/hip_runtime.h>
/* osgCompute - Copyright (C) 2008-2009 SVT Group
*                                                                     
* This library is free software; you can redistribute it and/or modify
* it under the terms of the GNU Lesser General Public License as
* published by the Free Software Foundation; either version 3 of
* the License, or (at your option) any later version.
*                                                                     
* This library is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of 
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU Lesse General Public License for more details.
*
* The full license is in LICENSE file included with this distribution.
*/

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
inline __device__ 
float lerp(float a, float b, float t)
{
    return a + t*(b-a);
}

//------------------------------------------------------------------------------
inline __device__ 
float4 operator+(float4 a, float4 b)
{
    return make_float4(a.x + b.x, a.y + b.y, a.z + b.z,  a.w + b.w); 
}


//------------------------------------------------------------------------------
inline __device__
float4 seed( float* seeds, unsigned int seedCount, unsigned int seedIdx, unsigned int ptclIdx, float3 bbmin, float3 bbmax )
{
    // random seed idx
    unsigned int idx1 = (seedIdx + ptclIdx) % seedCount;
    unsigned int idx2 = (idx1 + ptclIdx) % seedCount;
    unsigned int idx3 = (idx2 + ptclIdx) % seedCount;

    // seeds are within the range [0,1]
    float intFac1 = seeds[idx1];
    float intFac2 = seeds[idx2];
    float intFac3 = seeds[idx3];

    return make_float4(lerp(bbmin.x,bbmax.x,intFac1), lerp(bbmin.y,bbmax.y,intFac3),
        lerp(bbmin.z,bbmax.z,intFac2), 1);
}

//------------------------------------------------------------------------------
inline __device__
unsigned int thIdx()
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int width = gridDim.x * blockDim.x;

    return y*width + x;
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GLOBAL FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
__global__
void emitKernel( unsigned int numPtcls,
                   float4* ptcls, 
                   float* seeds, 
                   unsigned int seedIdx, 
                   float3 bbmin, 
                   float3 bbmax )
{
    // Receive particle pos
    unsigned int ptclIdx = thIdx();
    if( ptclIdx < numPtcls )
    {
        float4 curPtcl = ptcls[ptclIdx];

        // Reseed Particles if they
        // have moved out of the bounding box
        if( curPtcl.x < bbmin.x ||
            curPtcl.y < bbmin.y ||
            curPtcl.z < bbmin.z ||
            curPtcl.x > bbmax.x ||
            curPtcl.y > bbmax.y ||
            curPtcl.z > bbmax.z )
            ptcls[ptclIdx] = seed( seeds, numPtcls, seedIdx, ptclIdx, bbmin, bbmax );
    }
}

//------------------------------------------------------------------------------
__global__
void moveKernel( unsigned int numPtcls,
                 float4* ptcls, 
                 float etime )
{
    unsigned int ptclIdx = thIdx();
    if( ptclIdx < numPtcls )
    {
        // perform a euler step
        ptcls[ptclIdx] = ptcls[ptclIdx] + make_float4(0,0,etime,0);
    }
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS ////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
extern "C" __host__
void emit(unsigned int numPtcls, 
            void* ptcls, 
            void* seeds,  
            unsigned int seedIdx, 
            float3 bbmin, 
            float3 bbmax )
{
    dim3 blocks( (numPtcls / 128)+1, 1, 1 );
    dim3 threads( 128, 1, 1 );

    emitKernel<<< blocks, threads >>>(
        numPtcls,
        (float4*)ptcls,
        (float*)seeds,
        seedIdx,
        bbmin,
        bbmax );
}

//------------------------------------------------------------------------------
extern "C" __host__
void move( unsigned int numPtcls, 
           void* ptcls, 
           float etime )
{
    dim3 blocks( (numPtcls / 128)+1, 1, 1 );
    dim3 threads( 128, 1, 1 );

    moveKernel<<< blocks, threads >>>( 
        numPtcls,
        (float4*)ptcls,
        etime );
}