#include "hip/hip_runtime.h"
/* osgCompute - Copyright (C) 2008-2009 SVT Group
 *                                                                     
 * This library is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation; either version 3 of
 * the License, or (at your option) any later version.
 *                                                                     
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesse General Public License for more details.
 *
 * The full license is in LICENSE file included with this distribution.
*/

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
inline __device__ 
float lerp(float a, float b, float t)
{
    return a + t*(b-a);
}

//------------------------------------------------------------------------------
inline __device__
float4 reseed( float* seeds, unsigned int seedCount, unsigned int seedIdx, unsigned int ptclIdx, float3 bbmin, float3 bbmax )
{
    // random seed idx
    unsigned int idx1 = (seedIdx + ptclIdx) % seedCount;
    unsigned int idx2 = (idx1 + ptclIdx) % seedCount;
    unsigned int idx3 = (idx2 + ptclIdx) % seedCount;

    // seeds are within the range [0,1]
    float intFac1 = seeds[idx1];
    float intFac2 = seeds[idx2];
    float intFac3 = seeds[idx3];

    return make_float4(lerp(bbmin.x,bbmax.x,intFac1), lerp(bbmin.y,bbmax.y,intFac3),
                       lerp(bbmin.z,bbmax.z,intFac2), 1);
}

//------------------------------------------------------------------------------
inline __device__
unsigned int globalThreadIdx()
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int width = gridDim.x * blockDim.x;

    return y*width + x;
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GLOBAL FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
__global__
void k_reseed( float4* ptcls, float* seeds, unsigned int seedCount, unsigned int seedIdx, float3 bbmin, float3 bbmax )
{
    // compute particle idx
    unsigned int ptclIdx = globalThreadIdx();
    float4 curPtcl = ptcls[ptclIdx];

    // Reseed Particles if they
    // moved out of the bounding box
    if( curPtcl.x < bbmin.x ||
        curPtcl.y < bbmin.y ||
        curPtcl.z < bbmin.z ||
        curPtcl.x > bbmax.x ||
        curPtcl.y > bbmax.y ||
        curPtcl.z > bbmax.z )
        ptcls[ptclIdx] = reseed( seeds, seedCount, seedIdx, ptclIdx, bbmin, bbmax );
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS ////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
#include <osg/Vec4f>

//------------------------------------------------------------------------------
extern "C" __host__
void reseed( unsigned int numBlocks, 
             unsigned int numThreads, 
             void* ptcls, 
             void* seeds, 
             unsigned int seedCount, 
             unsigned int seedIdx, 
             osg::Vec3f bbmin, 
             osg::Vec3f bbmax )
{
    dim3 blocks( numBlocks, 1, 1 );
    dim3 threads( numThreads, 1, 1 );

    k_reseed<<< blocks, threads >>>(
                        reinterpret_cast<float4*>(ptcls),
                        reinterpret_cast<float*>(seeds),
                        seedCount,
                        seedIdx,
                        *reinterpret_cast<float3*>(&bbmin),
                        *reinterpret_cast<float3*>(&bbmax) );
}
