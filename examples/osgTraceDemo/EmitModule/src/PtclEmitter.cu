
#include <hip/hip_runtime.h>
/* osgCompute - Copyright (C) 2008-2009 SVT Group
*                                                                     
* This library is free software; you can redistribute it and/or modify
* it under the terms of the GNU Lesser General Public License as
* published by the Free Software Foundation; either version 3 of
* the License, or (at your option) any later version.
*                                                                     
* This library is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of 
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU Lesse General Public License for more details.
*
* The full license is in LICENSE file included with this distribution.
*/

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
inline __device__ 
float lerp(float a, float b, float t)
{
    return a + t*(b-a);
}

//------------------------------------------------------------------------------
inline __device__
float4 reseed( float* seeds, unsigned int seedCount, unsigned int seedIdx, unsigned int ptclIdx, float3 bbmin, float3 bbmax )
{
    // random seed idx
    unsigned int idx1 = (seedIdx + ptclIdx) % seedCount;
    unsigned int idx2 = (idx1 + ptclIdx) % seedCount;
    unsigned int idx3 = (idx2 + ptclIdx) % seedCount;

    // seeds are within the range [0,1]
    float intFac1 = seeds[idx1];
    float intFac2 = seeds[idx2];
    float intFac3 = seeds[idx3];

    return make_float4(lerp(bbmin.x,bbmax.x,intFac1), lerp(bbmin.y,bbmax.y,intFac3),
        lerp(bbmin.z,bbmax.z,intFac2), 1);
}

//------------------------------------------------------------------------------
inline __device__
unsigned int thIdx()
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int width = gridDim.x * blockDim.x;

    return y*width + x;
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GLOBAL FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
__global__
void reseedKernel( float4* ptcls, float* seeds, unsigned int seedCount, unsigned int seedIdx, float3 bbmin, float3 bbmax )
{
    // Receive particle pos
    unsigned int ptclIdx = thIdx();
    float4 curPtcl = ptcls[ptclIdx];

    // Reseed Particles if they
    // moved out of the bounding box
    if( curPtcl.x < bbmin.x ||
        curPtcl.y < bbmin.y ||
        curPtcl.z < bbmin.z ||
        curPtcl.x > bbmax.x ||
        curPtcl.y > bbmax.y ||
        curPtcl.z > bbmax.z )
        ptcls[ptclIdx] = reseed( seeds, seedCount, seedIdx, ptclIdx, bbmin, bbmax );
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS ////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
extern "C" __host__
void reseed(unsigned int numBlocks, 
            unsigned int numThreads, 
            void* ptcls, 
            void* seeds, 
            unsigned int seedCount, 
            unsigned int seedIdx, 
            float3 bbmin, 
            float3 bbmax )
{
    dim3 blocks( numBlocks, 1, 1 );
    dim3 threads( numThreads, 1, 1 );

    reseedKernel<<< blocks, threads >>>(
        reinterpret_cast<float4*>(ptcls),
        reinterpret_cast<float*>(seeds),
        seedCount,
        seedIdx,
        bbmin,
        bbmax );
}
