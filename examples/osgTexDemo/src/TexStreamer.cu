#include "hip/hip_runtime.h"
/* osgCompute - Copyright (C) 2008-2009 SVT Group
 *                                                                     
 * This library is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation; either version 3 of
 * the License, or (at your option) any later version.
 *                                                                     
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Lesse General Public License for more details.
 *
 * The full license is in LICENSE file included with this distribution.
*/

#ifndef TEXDEMO_TEXSTREAMER_KERNEL_H
#define TEXDEMO_TEXSTREAMER_KERNEL_H 1


texture<uchar4, 1, hipReadModeElementType> gaussTex; 
texture<uchar4, 2, hipReadModeNormalizedFloat> swapTex; 

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//-------------------------------------------------------------------------
__device__
float clamp( float value, float minValue, float maxValue )
{
    float erg = value;

    if( erg > maxValue )
        erg = maxValue;
    if( erg < minValue )
        erg = minValue;

    return erg;
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GLOBAL FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//-------------------------------------------------------------------------
__global__ 
void gaussKernel( uchar4* trg ) 
{
    // compute thread pos
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int width = gridDim.x * blockDim.x;

    int xPrev2 = ((x-2) < 0)? ((gridDim.x) * blockDim.x)-1 : x-2;
    int xPrev = ((x-1) < 0)? ((gridDim.x) * blockDim.x)-1 : x-1;
    int xNext = ((x+1) >= (gridDim.x * blockDim.x))? 0 : x+1;
    int xNext2 = ((x+2) >= (gridDim.x * blockDim.x))? 0 : x+2;

    int yPrev2 = ((y-2) < 0)? (gridDim.y * blockDim.y)-1 : y-2;
    int yPrev = ((y-1) < 0)? (gridDim.y * blockDim.y)-1 : y-1;
    int yNext = ((y+1) >= (gridDim.y * blockDim.y))? 0 : y+1;
    int yNext2 = ((y+2) >= (gridDim.y * blockDim.y))? 0 : y+2;

	// compute thread indices
    unsigned int idx[25];
    idx[0] = yPrev2 * width + xPrev2;
    idx[1] = yPrev2 * width + xPrev;
    idx[2] = yPrev2 * width + x;
    idx[3] = yPrev2 * width + xNext;
    idx[4] = yPrev2 * width + xNext2;
    idx[5] = yPrev * width + xPrev2;
    idx[6] = yPrev * width + xPrev;
    idx[7] = yPrev * width + x;
    idx[8] = yPrev * width + xNext;
    idx[9] = yPrev * width + xNext2;
    idx[10] = y* width + xPrev2;
    idx[11] = y* width + xPrev;
    idx[12] = y* width + x;
    idx[13] = y* width + xNext;
    idx[14] = y* width + xNext2;
    idx[15] = yNext* width + xPrev2;
    idx[16] = yNext* width + xPrev;
    idx[17] = yNext* width + x;
    idx[18] = yNext* width + xNext;
    idx[19] = yNext* width + xNext2;
    idx[20] = yNext2* width + xPrev2;
    idx[21] = yNext2* width + xPrev;
    idx[22] = yNext2* width + x;
    idx[23] = yNext2* width + xNext;
    idx[24] = yNext2* width + xNext2;

	// prepare weights
    float weights[25];
    weights[0] = 2;
    weights[1] = 7;
    weights[2] = 12;
    weights[3] = 7;
    weights[4] = 2;

    weights[5] = 7;
    weights[6] = 31;
    weights[7] = 52;
    weights[8] = 31;
    weights[9] = 7;

    weights[10] = 15;
    weights[11] = 52;
    weights[12] = 127;
    weights[13] = 52;
    weights[14] = 15;

    weights[15] = 7;
    weights[16] = 31;
    weights[17] = 52;
    weights[18] = 31;
    weights[19] = 7;

    weights[20] = 2;
    weights[21] = 7;
    weights[22] = 12;
    weights[23] = 7;
    weights[24] = 2;

	// perform gauss kernel
    float4 src = make_float4(0,0,0,0);
    for( unsigned int p=0; p<25; ++p )
    {
        uchar4 texValue = tex1Dfetch( gaussTex, idx[p] );
    
        src.x += weights[p] * texValue.x;
        src.y += weights[p] * texValue.y;
        src.z += weights[p] * texValue.z;
    }

    src.x /= 423.0f;
    src.y /= 423.0f;
    src.z /= 423.0f;

    src.x = clamp( src.x, 0, 255.0f );
    src.y = clamp( src.y, 0, 255.0f );
    src.z = clamp( src.z, 0, 255.0f );

	// write result
    trg[idx[12]] = 
		make_uchar4( 
			(unsigned char)(src.x), 
			(unsigned char)(src.y),
			(unsigned char)(src.z),
			255);
}

 
//-------------------------------------------------------------------------
__global__ 
void swapKernel( uchar4* trg ) 
{
    // compute thread dimension
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int width = gridDim.x * blockDim.x;
    unsigned int height = gridDim.y * blockDim.y;

    // compute target idx
    unsigned int trgIdx = y*width + x;

    // compute texture coordinates
    float2 texCoord = make_float2( ((float) x / (float) width) ,
                                   ((float) y / (float) height) );

    // sample value
    float4 src = tex2D( swapTex, texCoord.x, texCoord.y );
    // swap channels
    trg[trgIdx] = make_uchar4( 
                        (unsigned char)(src.z*255.0f), 
                        (unsigned char)(src.x*255.0f),
                        (unsigned char)(src.y*255.0f),
                        (unsigned char)(src.w*255.0f));
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS ////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//-------------------------------------------------------------------------
extern "C"
void gauss( const dim3& blocks, const dim3& threads, void* trg, void* src, unsigned int byteSize )
{
    hipError_t res = hipBindTexture( 0, gaussTex, src, byteSize ); 

    // call kernel
    gaussKernel<<< blocks, threads >>>( reinterpret_cast<uchar4*>(trg) );
}

//-------------------------------------------------------------------------
extern "C"
void swap( const dim3& blocks, const dim3& threads, void* trgBuffer, void* srcArray )
{
    // set texture parameters
    swapTex.normalized = true;                      // normalized texture coordinates (element of [0:1])
    swapTex.filterMode = hipFilterModeLinear;      // bilinear interpolation 
    swapTex.addressMode[0] = hipAddressModeClamp;  // wrap texture coordinates
    swapTex.addressMode[1] = hipAddressModeClamp;

    // bind texture
    hipError_t res = hipBindTextureToArray( swapTex, reinterpret_cast<hipArray*>(srcArray) );

    // call kernel
    swapKernel<<< blocks, threads >>>( reinterpret_cast<uchar4*>(trgBuffer) );
}

#endif // TEXDEMO_TEXSTREAMER_KERNEL_H